#include "hip/hip_runtime.h"
#include "table.cuh"
#include "board.cuh"

__host__ Table::Table(const size_t table_size) : size(table_size) {
  hipMallocManaged((void**)&entries, sizeof(Entry) * size);
  hipMallocManaged((void**)&mutex, sizeof(int) * size);
  hipMallocManaged((void**)&update_count, sizeof(ull));
  hipMallocManaged((void**)&hit_count, sizeof(ull));
  hipMallocManaged((void**)&lookup_count, sizeof(ull));
  *update_count = 0;
  *hit_count = 0;
  *lookup_count = 0;
  memset(entries, 0, sizeof(Entry) * size);
  memset(mutex, 0, sizeof(int) * size);
}

__host__ Table::~Table() {
#ifndef __CUDA_ARCH__
  if (enable) {
    hipFree(entries);
    hipFree(mutex);
    hipFree(update_count);
    hipFree(hit_count);
    hipFree(lookup_count);
  }
#endif
}

Table::Table(Table&& that)
  : entries(that.entries), mutex(that.mutex),
    update_count(that.update_count), hit_count(that.hit_count),
    lookup_count(that.lookup_count) {
  that.enable = false;
}

__device__ Entry Table::find(ull player, ull opponent) const {
  atomicAdd(lookup_count, 1);
  ull hash = (player + 17 * opponent) % size;
  Entry result;
  for (int i = 0; i < 32; ++i) {
    if (threadIdx.x % 32 == i) {
      lock(hash);
      result = entries[hash];
      if (result.player != player || result.opponent != opponent) {
        result.enable = false;
      } else if (result.enable) {
        atomicAdd(hit_count, 1);
      }
      unlock(hash);
    }
  }
  return result;
}

__device__ void Table::update(ull player, ull opponent, char upper, char lower, char value) const {
  if (upper <= lower) {
    return;
  }
  atomicAdd(update_count, 1);
  Entry entry;
  if (value > lower && value < upper) {
    entry = Entry(player, opponent, value, value);
  } else {
    if (value <= lower) {
      entry = Entry(player, opponent, value, -64);
    } else if (value >= upper) {
      entry = Entry(player, opponent, 64, value);
    } else {
      return;
    }
  }
  ull hash = (player + 17 * opponent) % size;
  for (int i = 0; i < 32; ++i) {
    if (threadIdx.x % 32 == i) {
      lock(hash);
      const Entry tmp = entries[hash];
      if (tmp.player != player || tmp.opponent != opponent || !tmp.enable) {
        entries[hash] = entry;
      } else {
        entries[hash].upper = min(tmp.upper, entry.upper);
        entries[hash].lower = max(tmp.lower, entry.lower);
      }
      unlock(hash);
    }
  }
}
